#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_vector_types.h>
#include <hip/hip_vector_types.h>
#include <stdlib.h>
//#include <hiprand.h>

#define NumThreadsX 16
#define NumThreadsY 16
#define NumThreadsZ 1
#define RenderWidth 1920
#define RenderHeight 1080
#define SamplesPerPixel 5000
#define InfinityBound 1e20

using namespace std;
// constants
__constant__ float3 white={1.0f, 1.0f, 1.0f}; 
__constant__ float3 black={0.0f,0.0f,0.0f};
__constant__ float3 Xaxis={1.0,0.0,0.0};
__constant__ float3 Yaxis={0.0,1.0,0.0};
__constant__ float3 OriginInit={50, 52, 295.6};
__constant__ float3 DirInitRaw={0, -0.042612, -1};
__constant__ float FoV=0.5135;
 
//some utility functions 
inline float clamp(float x)
{ 
	return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; 
} 

inline int RGBtoInt(float x)	  
{ 
	return int(pow(clamp(x), 1 / 2.2) * 255 + 0.5); 
}
//random number generator
__device__ static float RandGen(unsigned int *seed0, unsigned int *seed1) 
{
	*seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);  // hash the seeds using bitwise AND and bitshifts
	*seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

	unsigned int ires = ((*seed0) << 16) + (*seed1);	
	union 
	{
		float f;
	  	unsigned int ui;
	} res;

	res.ui = (ires & 0x007fffff) | 0x40000000;

	return (res.f - 2.f) / 2.f;	
}

//Define types of reflections
enum ReflectionType
{
	DIFF,
	SPEC,
	REFR
};
//Ray structure
struct Ray
{
	float3 Origin;
	float3 Direction;

	__device__ Ray(float3 orig_init, float3 dir_init) : Origin(orig_init), Direction(dir_init) {} 
};

//Sphere structure
struct Sphere
{
	float Radius;
	float3 Position,Emmisivity,Colour;
	ReflectionType Reflector;

	__device__ float SphereCollisionPoint(const Ray &inRay) const
	{
		float3 Distance=Position-inRay.Origin;
		float CollisionPoint,DecisionPoint=0.0001f;
		float b=dot(Distance,inRay.Direction);
		float Discriminant=b*b-dot(Distance,Distance)+Radius*Radius;
		if(Discriminant<0)
			return 0;
		else
			Discriminant=sqrtf(Discriminant);

		
		return	(CollisionPoint=b-Discriminant)>DecisionPoint?CollisionPoint:((CollisionPoint=b+Discriminant)>DecisionPoint?CollisionPoint:0);

	}
};

/* Scene definition 
{ float radius,
	{ float3 position },
	{ float3 emission },
	{ float3 colour },
	refl_type 
}
*/

__constant__ Sphere spheres[]=
{
	//Left 
	{ 1e5f,
		{ 1e5f + 1.0f, 40.8f, 81.6f },
		{ 0.0f, 0.0f, 0.0f },
		{ 0.75f, 0.25f, 0.25f },
		DIFF 
	}, 
	//Rght 
	{ 1e5f,
		{ -1e5f + 99.0f, 40.8f, 81.6f },
		{ 0.0f, 0.0f, 0.0f },
		{ .25f, .25f, .75f },
		DIFF 
	}, 
	//Back 
	{ 1e5f,
		{ 50.0f, 40.8f, 1e5f },
		{ 0.0f, 0.0f, 0.0f },
		{ .75f, .75f, .75f },
		DIFF 
	}, 
	//Front 
	{ 1e5f,
		{ 50.0f, 40.8f, -1e5f + 600.0f },
		{ 0.0f, 0.0f, 0.0f },
		{ 1.00f, 1.00f, 1.00f },
		DIFF 
	},
	//Bottom  
	{ 1e5f,
		{ 50.0f, 1e5f, 81.6f },
		{ 0.0f, 0.0f, 0.0f },
		{ .75f, .75f, .75f },
		DIFF 
	}, 	
	//Top
	{ 1e5f,
		{ 50.0f, -1e5f + 81.6f, 81.6f },
		{ 0.0f, 0.0f, 0.0f },
		{ .75f, .75f, .75f },
		DIFF 
	}, 
	// small sphere 1
	{ 16.5f,
		{ 27.0f, 16.5f, 47.0f },
		{ 12.0f, 12.4f, 12.2f },
		{ 0.999f, 0.999f, 0.999f},
		REFR
	}, 
	// small sphere 2
	{ 16.5f,
		{ 73.0f, 16.5f, 78.0f },
		{ 0.0f, 0.0f, 0.0f },
		{ 0.999f, 0.999f, 0.999f },
		SPEC 
	}, 
	// Light
	{ 600.0f,
		{ 50.0f, 681.6f - 0.27f, 81.6f },
		{ 12.0f, 12.0f, 12.0f },
		{ 0.9f, 0.2f, 0.086f },
		DIFF 
	}  

};


//Ray and scene intersection
__device__ inline bool DoesRayIntersectScene(const Ray &inRay, float &ClosestIntersection, int &HitID)
{
	int i;
	float SceneBlock=sizeof(spheres)/sizeof(Sphere);
	ClosestIntersection=InfinityBound;
	float NewClosestIntersection;
	for (i = int(SceneBlock); i--; )		
		if ((NewClosestIntersection=spheres[i].SphereCollisionPoint(inRay)) && NewClosestIntersection<ClosestIntersection)
		{
			ClosestIntersection=NewClosestIntersection;
			HitID=i;
		}
	
	return ClosestIntersection<InfinityBound;
		
}

__device__ float3 GetRadiance(Ray &inRay,unsigned int *seed1,unsigned int *seed2)
{

	float3 ColourAccumulator = black;
	float3 mask = white;
	
	int LightBounce;	
	
	for (LightBounce = 0; LightBounce < 4; ++LightBounce)		
	{
		float ClosestIntersection;
		int HitID=0;
		
		if (!DoesRayIntersectScene(inRay,ClosestIntersection,HitID))
				return black;

		const Sphere &HitObj=spheres[HitID];
		if (HitObj.Reflector==SPEC)		
		{
			
			float3 HitPoint=inRay.Origin+inRay.Direction*ClosestIntersection;
			float3 Normal=normalize(HitPoint-HitObj.Position);
			float3 FrontNormal=dot(Normal,inRay.Direction)<0 ? Normal : Normal*(-1); 

			ColourAccumulator+=mask*HitObj.Emmisivity;//*HitObj.Colour;

			float Azimuth = 2 * M_PI * RandGen(seed1, seed2);
			float Elevation = RandGen(seed1, seed2);
			float SqrtElev = sqrtf(Elevation); 
			float3 w = FrontNormal; 
			float3 u = normalize(cross((fabs(w.x) > 0.1 ? Yaxis : Xaxis), w));
			float3 v = cross(w,u);		
			float3 NewDir=normalize(u*cos(Azimuth)*SqrtElev + v*sin(Azimuth)*SqrtElev + w*sqrtf(1 - Elevation));																	

			inRay.Origin=HitPoint + FrontNormal*0.05f;
			inRay.Direction=NewDir;
			float3 UpdateMask=2*HitObj.Colour*dot(inRay.Direction-Normal*2*dot(Normal,inRay.Direction),inRay.Direction);
			mask *= UpdateMask; 		
			
		}
		else
		{
			float3 HitPoint=inRay.Origin+inRay.Direction*ClosestIntersection;
			float3 Normal=normalize(HitPoint-HitObj.Position);
			float3 FrontNormal=dot(Normal,inRay.Direction)<0 ? Normal : Normal*(-1); 

			ColourAccumulator+=mask*HitObj.Emmisivity;//*HitObj.Colour;

			float Azimuth = 2 * M_PI * RandGen(seed1, seed2);
			float Elevation = RandGen(seed1, seed2);
			float SqrtElev = sqrtf(Elevation); 
			float3 w = FrontNormal; 
			float3 u = normalize(cross((fabs(w.x) > 0.1 ? Yaxis : Xaxis), w));
			float3 v = cross(w,u);		
			float3 NewDir=normalize(u*cos(Azimuth)*SqrtElev + v*sin(Azimuth)*SqrtElev + w*sqrtf(1 - Elevation));																	

			inRay.Origin=HitPoint + FrontNormal*0.05f;
			inRay.Direction=NewDir;
			float3 UpdateMask=2*HitObj.Colour*dot(NewDir,FrontNormal);
			mask *= UpdateMask; 		
		}
		
		
		
	}
		
		
	

	return ColourAccumulator;
}

__global__ void TracePath2(float3 *RenderedImage)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int CurrentPixel = (RenderHeight - y - 1)*RenderWidth + x; 

	unsigned int seed1 = x;
	unsigned int seed2 = y;
	
	Ray CameraRay(OriginInit,normalize(DirInitRaw));
	float3 DirOffsetX=make_float3(RenderWidth*FoV/RenderHeight,0.0f,0.0f);
	float3 DirOffsetY = normalize(cross(DirOffsetX, CameraRay.Direction)) * FoV;
	float3 FinalPixCol = black;

	for (int CurrentSample = 0; CurrentSample < SamplesPerPixel; CurrentSample++)
	{		
		float3 DirectionOffset = CameraRay.Direction + DirOffsetX*((0.25 + x) / RenderWidth - 0.5) + DirOffsetY*((0.25 + y) / RenderHeight - 0.5);		
		Ray temp_ray(CameraRay.Origin + DirectionOffset * 40, normalize(DirectionOffset));		
		FinalPixCol+=GetRadiance(temp_ray, &seed1, &seed2)*(1.0 / SamplesPerPixel); 
	}

	RenderedImage[CurrentPixel]=make_float3(clamp(FinalPixCol.x, 0.0f, 1.0f), clamp(FinalPixCol.y, 0.0f, 1.0f), clamp(FinalPixCol.z, 0.0f, 1.0f));
}

int main(int argc, char const *argv[])
{
	float3* h_RenderedImage = new float3[RenderWidth*RenderHeight*sizeof(float3)]; 
	float3* d_RenderedImage;    
	int PixPtr;	

	

	hipMalloc(&d_RenderedImage, RenderWidth * RenderHeight * sizeof(float3));
	
	dim3 block(NumThreadsX,NumThreadsY,NumThreadsZ);   
	dim3 grid(RenderWidth / NumThreadsX, RenderHeight / NumThreadsY, NumThreadsZ);

	printf("Starting Path Trace Kernel\n");
	TracePath2 <<< grid,block>>> (d_RenderedImage);	
	hipMemcpy(h_RenderedImage, d_RenderedImage, RenderWidth * RenderHeight *sizeof(float3), hipMemcpyDeviceToHost);  	
	hipFree(d_RenderedImage);  
	printf("Finished and freed\n");

	FILE *f = fopen("GPU_image.ppm", "w");          
	fprintf(f, "P3\n%d %d\n%d\n", RenderWidth, RenderHeight, 255);

	for (PixPtr = 0; PixPtr < RenderWidth*RenderHeight; PixPtr++)  
		fprintf(f, "%d %d %d ", RGBtoInt(h_RenderedImage[PixPtr].x),RGBtoInt(h_RenderedImage[PixPtr].y),RGBtoInt(h_RenderedImage[PixPtr].z));

	printf("Saved image\n");

	delete[] h_RenderedImage;
	
	return 0;
}